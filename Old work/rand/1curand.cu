
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define SCALE 1023
#define DSIZE 5000
#define nTPB 256

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

__device__ float getnextrand(hiprandState *state){

  return (float)(hiprand_uniform(state));
}

__device__ int getnextrandscaled(hiprandState *state, int scale){

  return (int) scale * getnextrand(state);
}


__global__ void initCurand(hiprandState *state, unsigned long seed){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, 0, 0, &state[idx]);
}

__global__ void testrand(hiprandState *state, float *a1, float *a2){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

//    a1[idx] = getnextrandscaled(&state[idx], SCALE);
//    a2[idx] = getnextrandscaled(&state[idx], SCALE);
    a1[idx] = getnextrand(&state[idx]);
    a2[idx] = getnextrand(&state[idx]);

}

int main() {

    float *h_a1, *h_a2, *d_a1, *d_a2;
    hiprandState *devState;

    h_a1 = (float *)malloc(DSIZE*sizeof(float));
    if (h_a1 == 0) {printf("malloc fail\n"); return 1;}
    h_a2 = (float *)malloc(DSIZE*sizeof(float));
    if (h_a2 == 0) {printf("malloc fail\n"); return 1;}
    hipMalloc((void**)&d_a1, DSIZE * sizeof(float));
    hipMalloc((void**)&d_a2, DSIZE * sizeof(float));
    hipMalloc((void**)&devState, DSIZE * sizeof(hiprandState));
    cudaCheckErrors("cudamalloc");



     initCurand<<<(DSIZE+nTPB-1)/nTPB,nTPB>>>(devState, 1);
     hipDeviceSynchronize();
     cudaCheckErrors("kernels1");
     testrand<<<(DSIZE+nTPB-1)/nTPB,nTPB>>>(devState, d_a1, d_a2);
     hipDeviceSynchronize();
     cudaCheckErrors("kernels2");
     hipMemcpy(h_a1, d_a1, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
     hipMemcpy(h_a2, d_a2, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
     cudaCheckErrors("cudamemcpy");
     printf("1st returned random value is %f\n", h_a1[0]);
     printf("2nd returned random value is %f\n", h_a2[0]);

     for (int i=1; i< DSIZE; i++){
       if (h_a1[i] != h_a1[0]) {
         printf("mismatch on 1st value at %d, val = %f\n", i, h_a1[i]);
         return 1;
         }
       if (h_a2[i] != h_a2[0]) {
         printf("mismatch on 2nd value at %d, val = %f\n", i, h_a2[i]);
         return 1;
         }
       }
     printf("thread values match!\n");

return 0;
}


