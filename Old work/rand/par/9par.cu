//#include<conio.h>

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <sys/time.h>
#include "time.h"

using namespace std;

//struct timeval tv;


__global__ void parMap(float *pD, float *netD, int grid)
{
        unsigned int rID= blockDim.x*blockIdx.x + threadIdx.x;
        int left, right, top, bottom;
        float x,y, fL,fR,fB,fT;


        //x=pD[1];
        x = pD[rID*2];
        //x=102.358000;

        y = pD[rID*2+1];
                                //y=320.568000;
                                //printf("%d %f %f       ",rID,x,y);

        //printf("thread: %d x:%f, y:%f  \n", rID,x,y);

        left = (int)floorf(x);
        right = left + 1;

        bottom = (int)floorf(y);
        top = bottom +1;

        //printf("left:%d, right:%d,top:%d, bottom:%d   \n", left, right, top, bottom );

        if (left>= grid||right>= grid||top>= grid||bottom>= grid)
                {
                        left=0;
                        right=1;
                        top=1;
                        bottom = 0;
                        x=0.500000;
                        y=0.500000;
                }

                fL = x - left;
                fR = 1 - fL;

                fB = y - bottom;
                fT = 1 - fB;

        //      printf("fL:%f, fR:%f, fT:%f, fB:%f L:%d, R:%d, T:%d, B:%d   \n", fL, fR, fT,fB, left, right, top, bottom );

        //      printf("L:%d, R:%d, T:%d, B:%d   \n", left, right, top, bottom );


        //      printf("grid: left:%f, right:%f, top:%f, bottom:%f \n", netD[left], netD[right], netD[top], netD[bottom]);
                netD[grid*left + bottom]  = netD[grid*left + bottom] +(fT*fR);
                netD[grid*right + bottom] = netD[grid*right + bottom]+(fT*fL);
                netD[grid*left+ top]      = netD[grid*left + top]    +(fB*fR);
                netD[grid*right+ top]     = netD[grid*right + top]   +(fB*fL);

//              if(rID%50==1)
//                      printf("grid: left:%f, right:%f, top:%f, bottom:%f \n", netD[left], netD[right], netD[top], netD[bottom]);


}



// main function
int main(int argc, char *argv[])
{
        int grid = 1024, i, j, lp=1,max = grid, sizeGrid= grid*grid;
        unsigned int par = 160000, loop=2000, sizePar = 2*par;

        hipEvent_t s_i, e_i, s_mc_h2d, e_mc_h2d, s_mc_d2h, e_mc_d2h, s_pl, e_pl;
        float t_i=0.0, t_mc_h2d=0.0, t_mc_d2h=0.0, t_pl=0.0, ti=0.0, tmc_h2d=0.0, tpl=0.0;

        hipEventCreate(&s_i);          hipEventCreate(&s_mc_h2d);
        hipEventCreate(&e_i);          hipEventCreate(&e_mc_h2d);

        hipEventCreate(&s_mc_d2h);             hipEventCreate(&s_pl);
        hipEventCreate(&e_mc_d2h);             hipEventCreate(&e_pl);




        /*
                float* netH;
                float* pH;
                float* netD;
                float*  pD;
                */

        float *netH, *pH, *netD,  *pD;

        hipEventRecord(s_i,0);

        netH =        (float*)malloc(sizeof(float)*sizeGrid);
        pH   =        (float*)malloc(sizeof(float)*sizePar);

        //intialising particles.


        hipEventRecord( s_i,0 );

        for(i=0;i< grid;i++)
                for(j=0;j< grid;j++)
                        netH[grid*i+j]=0.0;
        for( i = 0; i < sizePar; i++)
                pH[i]= ((float)rand()/(float)(RAND_MAX) * (float)(max-1));


        hipMalloc( (void **)&netD, sizeof(float)*sizeGrid);
        hipMemcpy(netD, netH, sizeGrid*(sizeof(float)),  hipMemcpyHostToDevice);

                hipEventRecord( e_i,0 );
                hipEventSynchronize( e_i );
                hipEventElapsedTime( &ti, s_i, e_i);


	for(lp=1;lp<loop;lp++)
	{
        //	for( i = 0; i < sizePar; i++)
        //	       pH[i]= ((float)rand()/(float)(RAND_MAX) * (float)(max-1));

//        printf("particle initialised \n ");
//        printf("Grid initialised \n ");
/*       	cudaEventRecord( e_i,0 );
        	cudaEventSynchronize( e_i );
        	cudaEventElapsedTime( &ti, s_i, e_i);
*/
        // Allocating GPU memory
        	hipEventRecord(s_mc_h2d,0);
//        cudaMalloc( (void **)&netD, sizeof(float)*sizeGrid);
        	hipMalloc( (void **)&pD, sizeof(float)*sizePar);
//        printf("Cuda memory allocated \n ");
        //transfering data to gpu
        	hipMemcpy( pD,   pH,   sizePar*(sizeof(float)),  hipMemcpyHostToDevice);
//        cudaMemcpy(netD, netH, sizeGrid*(sizeof(float)),  cudaMemcpyHostToDevice);

        	hipEventRecord( e_mc_h2d,0 );
        	hipEventSynchronize( e_mc_h2d );
        	hipEventElapsedTime( &tmc_h2d, s_mc_h2d, e_mc_h2d);
//        printf("Data cpy to gpu \n \n ");
        //initialising the thread in groups
        	hipEventRecord( s_pl,0 );
        	dim3 dimBlock(192);
        	dim3 dimGrid((par/192));
//        printf("Thread launched \n \n ");
        //@@ Launch the GPU Kernel here
	        parMap<<<dimGrid, dimBlock>>>(pD, netD, grid);
//        printf("Data back to CPU \n \n ");
	        hipEventRecord( e_pl,0 );
	        hipEventSynchronize( e_pl );
	        hipEventElapsedTime( &tpl, s_pl, e_pl);

		t_i+=ti;
		t_mc_h2d+=tmc_h2d;
		t_pl+=tpl;	
	}


        // Copy the results in GPU memory back to the CPU
        hipEventRecord( s_mc_d2h,0 );

        hipMemcpy(netH, netD, sizeof(float)*sizeGrid, hipMemcpyDeviceToHost);

        hipEventRecord( e_mc_d2h,0 );
        hipEventSynchronize( e_mc_d2h );
        hipEventElapsedTime( &t_mc_d2h, s_mc_d2h, e_mc_d2h);



        FILE *f = fopen("file.txt", "w");
	par*=loop;
        if (f == NULL)
        {
            printf("Error opening file!\n");
            exit(1);
        }


        float avg= par/(max*max);



        for ( i = 0; i < sizeGrid; ++i)
        {
                        //cout<<netH[i]<<" ";
        fprintf (f,"%f ",((netH[i])/avg))  ;// /temp1));
            if (i%grid==(grid-1))
                {
                        //printf("\n");
                        fprintf (f," \n" );
                }
        }

        fclose(f);

//        cout<<"Grid size: "<<grid<<"x"<<grid<<"  particles:"<<par <<"\n";
//        cout<<"Initialisation time: "<<t_i<<"\n";
//        cout<<"Memory copy H 2 d:   "<<t_mc_h2d<<"\n";
//        cout<<"Memory copy D 2 H:   "<<t_mc_d2h<<"\n";
//        cout<<"Processing time:     "<<t_pl<<"\n";
//        cout<<"Total time:          "<<( t_mc_h2d + t_mc_d2h + t_pl )<<"\n";



       printf("\n\nGrid size: \t\t%d \nparticle:\t %d\n", grid,par);
       printf("\nInitialisation time:\t%f \n", t_i);
       printf("\nMemory Copy H 2 D:\t%f \n", t_mc_h2d);
       printf("\nMemory Copy D 2 H:\t%f \n", t_mc_d2h);
       printf("\nProcessing time:\t%f \n\n", t_pl);




        //event destroy
        hipEventDestroy(s_i);    hipEventDestroy(s_mc_h2d);    hipEventDestroy(s_mc_d2h);
        hipEventDestroy(e_i);    hipEventDestroy(e_mc_h2d);    hipEventDestroy(e_mc_d2h);
        hipEventDestroy(s_pl);
        hipEventDestroy(e_pl);


        // Free the GPU memory
        hipFree(netD);
        hipFree(pD);

        free(netH);
        free(pH);

        return 0;
}

