/* Algo to so the weight distribution of 5000 particle on a
   grid of 64x64 */


//#include<conio.h>

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <sys/time.h>

using namespace std;


/*#define funcCheck(stmt) do {                                                    \
        cudaError_t err = stmt;                                               \
        if (err != cudaSuccess) {                                             \
            printf( "Failed to run stmt %d ", __LINE__);                       \
            printf( "Got CUDA error ...  %s \n", cudaGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)*/


//__device__ float floorf       (float  x);


__global__ void parMap(float *pD, float *netD, int grid)
{
        unsigned int rID= blockDim.x*blockIdx.x + threadIdx.x;
        int left, right, top, bottom;
        float x,y, fL,fR,fB,fT;


        //x=pD[1];
        x = pD[rID*2];
        //x=102.358000;

        y = pD[rID*2+1];
                                //y=320.568000;
                                //printf("%d %f %f       ",rID,x,y);

        //printf("thread: %d x:%f, y:%f  \n", rID,x,y);

        left = (int)floorf(x);
        right = left + 1;

        bottom = (int)floorf(y);
        top = bottom +1;

        //printf("left:%d, right:%d,top:%d, bottom:%d   \n", left, right, top, bottom );

        if (left>= grid||right>= grid||top>= grid||bottom>= grid)
                {
                        left=0;
                        right=1;
                        top=1;
                        bottom = 0;
                        x=0.500000;
                        y=0.500000;
                }

                fL = x - left;
                fR = 1 - fL;

                fB = y - bottom;
                fT = 1 - fB;

        //      printf("fL:%f, fR:%f, fT:%f, fB:%f L:%d, R:%d, T:%d, B:%d   \n", fL, fR, fT,fB, left, right, top, bottom );

        //      printf("L:%d, R:%d, T:%d, B:%d   \n", left, right, top, bottom );


        //      printf("grid: left:%f, right:%f, top:%f, bottom:%f \n", netD[left], netD[right], netD[top], netD[bottom]);
                netD[grid*left + bottom]  = netD[grid*left + bottom] +(fT*fR);
                netD[grid*right + bottom] = netD[grid*right + bottom]+(fT*fL);
                netD[grid*left+ top]      = netD[grid*left + top]    +(fB*fR);
                netD[grid*right+ top]     = netD[grid*right + top]   +(fB*fL);

                printf("grid: left:%f, right:%f, top:%f, bottom:%f \n", netD[left], netD[right], netD[top], netD[bottom]);


}



// main function
int main(int argc, char *argv[])
{
        int grid = 1024, i, j, max = grid, sizeGrid= grid*grid;
        unsigned int par = 850000, sizePar = 2*par;

        hipEvent_t s_i, e_i, s_mc_h2d, e_mc_h2d, s_mc_d2h, e_mc_d2h, s_pl, e_pl;
        float t_i, t_mc_h2d, t_mc_d2h, t_pl;

        hipEventCreate(&s_i);          hipEventCreate(&s_mc_h2d);
        hipEventCreate(&e_i);          hipEventCreate(&e_mc_h2d);

        hipEventCreate(&s_mc_d2h);             hipEventCreate(&s_pl);
        hipEventCreate(&e_mc_d2h);             hipEventCreate(&e_pl);




        float *netH, *pH, *netD,  *pD;

        hipEventRecord(s_i,0);

        netH =        (float*)malloc(sizeof(float)*sizeGrid);
        pH   =        (float*)malloc(sizeof(float)*sizePar);
        //intialising particles.

        for( i = 0; i < sizePar; i++)
               pH[i]= ((float)rand()/(float)(RAND_MAX) * (float)max);

        //printf("particle initialised \n ");


        for(i=0;i< grid;i++)
                for(j=0;j< grid;j++)
                        netH[grid*i+j]=0.0;

        //printf("Grid initialised \n ");
        hipEventRecord( e_i,0 );
        hipEventSynchronize( e_i );
        hipEventElapsedTime( &t_i, s_i, e_i);

        // Allocating GPU memory
        hipEventRecord(s_mc_h2d,0);
        hipMalloc( (void **)&netD, sizeof(float)*sizeGrid);
        hipMalloc( (void **)&pD, sizeof(float)*sizePar);

                //printf("Cuda memory allocated \n ");

        //transfering data to gpu
        hipMemcpy( pD,   pH,   sizePar*(sizeof(float)),  hipMemcpyHostToDevice);
        hipMemcpy(netD, netH, sizeGrid*(sizeof(float)),  hipMemcpyHostToDevice);

        hipEventRecord( e_mc_h2d,0 );
        hipEventSynchronize( e_mc_h2d );
        hipEventElapsedTime( &t_mc_h2d, s_mc_h2d, e_mc_h2d);

                //printf("Data cpy to gpu \n \n ");

        //initialising the thread in groups
        hipEventRecord( s_pl,0 );
        dim3 dimBlock(192);
        dim3 dimGrid((par/192));

        //@@ Launch the GPU Kernel here
        parMap<<<dimGrid, dimBlock>>>(pH, netH, grid);
                //printf("Data back to CPU \n \n ");

        hipEventRecord( e_pl,0 );
        hipEventSynchronize( e_pl );
        hipEventElapsedTime( &t_pl, s_pl, e_pl);



        // Copy the results in GPU memory back to the CPU
        hipEventRecord( s_mc_d2h,0 );

        hipMemcpy(netH, netD, sizeof(float)*sizeGrid, hipMemcpyDeviceToHost);

        hipEventRecord( e_mc_d2h,0 );
        hipEventSynchronize( e_mc_d2h );
        hipEventElapsedTime( &t_mc_d2h, s_mc_d2h, e_mc_d2h);


//!! if(x<0) stop print i
//!! denominator -- nan


        FILE *f = fopen("file.txt", "w");
        if (f == NULL)
        {
            printf("Error opening file!\n");
            exit(1);
        }

        //float temp1=par/(sizeGrid);

        for ( i = 0; i < sizeGrid; ++i)
        {
                        //cout<<netH[i]<<" ";
        fprintf (f,"%f ",((netH[i])))  ;// /temp1));
            if (i%grid==(grid-1))
                {
                        //printf("\n");
                        fprintf (f," \n" );
                }
        }

        fclose(f);

        cout<<"Grid size: "<<grid<<"x"<<grid<<"  particles:"<<par <<"\n";
        cout<<"Initialisation time: "<<t_i<<"\n";
        cout<<"Memory copy H 2 d:   "<<t_mc_h2d<<"\n";
        cout<<"Memory copy D 2 H:   "<<t_mc_d2h<<"\n";
        cout<<"Processing time:     "<<t_pl<<"\n";
        cout<<"Total time:          "<<( t_mc_h2d + t_mc_d2h + t_pl )<<"\n";

        //event destroy
        hipEventDestroy(s_i);
        hipEventDestroy(s_mc_h2d);
        hipEventDestroy(s_mc_d2h);
        hipEventDestroy(e_i);
        hipEventDestroy(e_mc_h2d);
        hipEventDestroy(e_mc_d2h);
        hipEventDestroy(s_pl);
        hipEventDestroy(e_pl);


        // Free the GPU memory
        hipFree(netD);
        hipFree(pD);

        free(netH);
        free(pH);

        return 0;
}
