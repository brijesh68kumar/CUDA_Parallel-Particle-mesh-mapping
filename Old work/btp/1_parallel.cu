#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

int main(void) {
    void MatrixMultiplication(float *, float *, int, int);
    const int Width = 64;
	const int nP = 50000;
    float M[nP*2];      //, N[Width*Width], 
	float P[Width*Width];
    for(int i = 0; i < (nP*2) ; i++) {
        M[i] = ((float)rand()/(float)(RAND_MAX) * (float)Width);;
    }
	
    MatrixMultiplication(M, P, Width, nP);
    for(int i = 0; i < (Width*Width) ; i++) {
        printf("%f \n", P[i]);
    }
	
	
	
	FILE *f = fopen("Par.txt", "w");
	if (f == NULL)
	{
    	printf("Error opening file!\n");
    	exit(1);
	}
		int i;
		for ( i = 0; i < Width*Width ; ++i)
		{

			fprintf (f,"%f ,",P[i] );
			if ((i %Width) ==0)
			{
				fprintf (f,"\n" );
			}
		}

/*
	int i = 1;
	float py = 3.1415927;
	fprintf(f, "Integer: %d, float: %f\n", i, py);
*/
	fclose(f);

	
    int quit;
    scanf("%d",&quit);
    return 0;
}

//Matrix multiplication kernel - thread specification
__global__ void MatrixMulKernel(float *Md, float *net, int Width, int numPar) {
    //2D Thread ID
    int tx = threadIdx.x;
    //    int ty = threadIdx.y;
	

    //Pvalue stores the Pd element that is computed by the thread
    //float Pvalue = 0;
  /*  for(int k = 0; k < Width ; ++k) {
        float Mdelement = Md[ty * Width + k];
     //   float Ndelement = Nd[k*Width + tx];
        Pvalue += (Mdelement*Mdelement);
    }	*/
	if (tx%2 == 0)
		{
		
	
		float x = Md[tx];
		float y = Md[tx+1];

		int left = (int)floor(x);
		int right = left + 1;

		int bottom = (int)floor(y);
		int top = bottom +1;

			float fL = x - left;
			float fR = 1 - fL;

			float fB = y - bottom;
			float fT = 1 - fB;

			net[left*Width+ bottom] 	= 	net[left*Width+ bottom]	+( fT * fR ) ;
			net[right*Width+ bottom] 	= 	net[right*Width+ bottom]	+( fT * fL ) ;
			net[left*Width+ top]		=	net[left*Width+ top]		+( fB * fR ) ;
			net[right*Width+ top]		=	net[right*Width+ top]		+( fB * fL ) ;

	
	
		}
    //Pd[ty*Width + tx] = Pvalue;
}





void MatrixMultiplication(float *M, float *P, int Width, int numPar ) {
    int size = Width*Width*sizeof(float);
    float *Md;
	float *Pd;
	int sizep = 2*numPar*sizeof(float);

    //Transfer M and N to device memory
    hipMalloc((void**)&Md, sizep);
    hipMemcpy(Md,M,sizep,hipMemcpyHostToDevice);
  
    //cudaMalloc((void**)&Nd, size);
    //cudaMemcpy(Nd,N,size,cudaMemcpyHostToDevice);

    //Allocate P on the device
    hipMalloc((void**)&Pd,size);

    //Setup the execution configuration
    dim3 dimBlock(Width,Width);
    dim3 dimGrid(1,1);

    //Launch the device computation threads!
    MatrixMulKernel<<<dimGrid,dimBlock>>>(Md,Pd,Width,numPar);

    //Transfer P from device to host
    hipMemcpy(P,Pd,size,hipMemcpyDeviceToHost);

    //Free device matrices
    hipFree(Md);
//    cudaFree(Nd);
    hipFree(Pd);
}
