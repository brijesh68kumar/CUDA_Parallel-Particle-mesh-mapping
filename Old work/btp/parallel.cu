#include "hip/hip_runtime.h"
#include<iostram>
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

#define fram 64
#define par 50000

__global__ int parMesh(float d_p[par][2] , int d_net[fram][fram] )
{
	int i = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	if(i >= par|| col>=2)
	{
	    return 0 ;
	}
	else
	{

	    float x = d_p[i][0];
	    float y = d_p[i][1];
	    int left = (int)floor(x);
	    int right = left + 1;
	    int bottom = (int)floor(y);
	    int top = bottom +1;

	//if ((left<=32)&&(top<=32))
	
		/* code */
		
	    float fL = x - left;
	    float fR = 1 - fL;

	    float fB = y - bottom;
	    float fT = 1 - fB;

	    		d_net[left][bottom]	= net[left][bottom]+( fT * fR ) ;
			d_net[right][bottom]    = net[right][bottom]	+ ( fT * fL ) ;
			d_net[left][top]	= net[left][top]     + ( fB * fR ) ;
			d_net[right][top]	= net[right][top]	+ ( fB * fL ) ;
	}
}


int main()
{
       //Writing the results in a file 


	FILE *f = fopen("parallelCuda.txt", "w");
	if (f == NULL)
	{
    	    printf("Error opening file!\n");
    	    exit(1);
	}
	for ( i = 0; i < 64; ++i)
	{
	    for ( j = 0; j < 64; j++)
	    {
		fprintf (f,"%f ,",net[i][j] );
   	    }
	    fprintf (f,"\n" );
	}
	fclose(f);
}
