#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

//Matrix multiplication kernel - thread specification
__global__ void MatrixMulKernel(float *Md, float *net, int Width, int numPar) {
	//2D Thread ID
	int tx = threadIdx.x;

	if (tx % 2 == 0)
	{


		float x = Md[tx];
		float y = Md[tx + 1];

//		printf("hello");

		int left = (int)floor(x);
		int right = left + 1;

		int bottom = (int)floor(y);
		int top = bottom + 1;

		float fL = x - left;
		float fR = 1 - fL;

		float fB = y - bottom;
		float fT = 1 - fB;

		net[left*Width + bottom] = net[left*Width + bottom] + (fT * fR);
		net[right*Width + bottom] = net[right*Width + bottom] + (fT * fL);
		net[left*Width + top] = net[left*Width + top] + (fB * fR);
		net[right*Width + top] = net[right*Width + top] + (fB * fL);
	}
	
}





void MatrixMultiplication(float *M, float *P, int Width, int numPar) {
	int size = Width*Width*sizeof(float);
	float *Md;
	float *Pd;
	int sizep = 2 * numPar*sizeof(float);

	//Transfer M and N to device memory
	hipMalloc((void**)&Md, sizep);
	hipMemcpy(Md, M, sizep, hipMemcpyHostToDevice);
	printf("before allocation");
	//Allocate P on the device
	hipMalloc((void**)&Pd, size);

	//Setup the execution configuration
	dim3 dimBlock(Width, Width);
	dim3 dimGrid(1, 1);

	//Launch the device computation threads!
	MatrixMulKernel << <dimGrid, dimBlock >> >(Md, Pd, Width, numPar);

	printf("after kernal");

	//Transfer P from device to host
	hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);

	//Free device matrices
	hipFree(Md);
	hipFree(Pd);
}



int main(void) {
	void MatrixMultiplication(float *, float *, int, int);
	const int Width = 64;
	const int nP = 50000;
	float M[nP * 2];      //, N[Width*Width], 
	float P[Width*Width];
	for (int i = 0; i < (nP * 2); i++) {
		M[i] = ((float)rand() / (float)(RAND_MAX)* (float)Width);;
	}

	MatrixMultiplication(M, P, Width, nP);
	for (int i = 0; i < (Width*Width); i++) {
//		printf("%f \n", P[i]);
	}



	FILE *f = fopen("Par.txt", "w");
	if (f == NULL)
	{
		printf("Error opening file!\n");
		exit(1);
	}

	/*
	int i;
	for (i = 0; i < Width*Width; ++i)
	{

		fprintf(f, "%f ,", P[i]);
		if ((i %Width) == 0)
		{
			fprintf(f, "\n");
		}
	}

	fclose(f);
	*/

	int i;
	for (i = 0; i < Width; ++i)
	{

		printf("%f ,", P[i]);
		if ((i %Width) == 0)
		{
			printf( "\n");
		}
	}




	int quit;
	scanf("%d", &quit);
	return 0;
}


