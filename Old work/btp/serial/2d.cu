
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
 
int main()
{
    int r = 100, c = 2,out=100, i, j, count,max=1024, loop, left, right, top, bottom;
    float net[1024][1024], x, y, fL, fR, fB, fT, time, totalTime;

	hipEvent_t start1, stop1;
        hipEventCreate(&start1);
        hipEventCreate(&stop1);
  


    

 
    float **arr = (float **)malloc(r * sizeof(float *));
    for (i=0; i<r; i++)
         arr[i] = (float *)malloc(c * sizeof(float));
 

    for (loop=0;loop<out;loop++)
        {
    // Note that arr[i][j] is same as *(*(arr+i)+j)
        for (i = 0; i <  r; i++)
          for (j = 0; j < c; j++)
             arr[i][j] =((float)rand()/(float)(RAND_MAX) * (float)max) ;  // OR *(*(arr+i)+j) = ++count
 
	hipEventRecord (start1,0);

        for (i = 0; i <  r-1 ; i++)
	    {
//          for (j = 0; j < c; j++)
//             printf("%f ", arr[i][j]);

                x = arr[i][0];
                y = arr[i][1];

                left = (int)floor(x);
                right = left + 1;

                bottom = (int)floor(y);
                top = bottom +1;

                fL = x - left;
                fR = 1 - fL;

                fB = y - bottom;
                fT = 1 - fB;

		if (left>=1024||right>=1024||bottom>=1024||top>=1024)		break;

                net[left][bottom]  = net[left][bottom]  +( fT*fR );
                net[right][bottom] = net[right][bottom] +( fT*fL );
                net[left][top]     = net[left][top]     +( fB*fR );
                net[right][top]    = net[right][top]    +( fB*fL );
	    }
                hipEventRecord (stop1,0);
                hipEventSynchronize ( stop1 );
                hipEventElapsedTime ( &time, start1, stop1 );
                printf("\n seconds: %f \n", time);

                hipEventDestroy(start1);
                hipEventDestroy(stop1);

                totalTime = totalTime+time;
        }

        printf("\nSeconds: %f \n", totalTime);
                hipEventDestroy(start1);
                hipEventDestroy(stop1);


   /* Code for further processing and free the 
      dynamically allocated memory */

        FILE *f = fopen("file.txt", "w");
        if (f == NULL)
        {
        printf("Error opening file!\n");
        exit(1);
        }

                for ( i = 0; i < 1023; ++i)
                {
                        for ( j = 0; j < 1023; j++)
                        {

                                fprintf (f,"%f ,",net[i][j] );
                        }

                        fprintf (f,"\n" );
                }

        fclose(f);




 
   return 0;
}

