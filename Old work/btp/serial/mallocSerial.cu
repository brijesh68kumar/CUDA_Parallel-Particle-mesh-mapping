/* Algo to so the weight distribution of 5000 particle on a
   grid of 64x64 */


//#include<conio.h>

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>



// main function
int main(int argc, char *argv[])
{

//        int numElement = 2;
        int numArray   = 1000;
//        int loop       = 1000;

        int i;
        int j;
        int max = 1024;
        float net[1024][1024];

        float **ptr;
        ptr = (float **)malloc(numArray*sizeof(float));


        for(i=0; i < numArray; i++)
        {
         *(*(ptr+i)+0) = ((float)rand()/(float)(RAND_MAX) * (float)max);
         *(*(ptr+i)+1) = ((float)rand()/(float)(RAND_MAX) * (float)max);
        }

        for(i=0; i < numArray; i++)
        {
                float x = *(*(ptr+i)+0);
                float y = *(*(ptr+i)+1);

                int left = (int)floor(x);
                int right = left + 1;

                int bottom = (int)floor(y);
                int top = bottom +1;

                float fL = x - left;
                float fR = 1 - fL;

                float fB = y - bottom;
                float fT = 1 - fB;

                net[left][bottom]  = net[left][bottom]  +( fT*fR );
                net[right][bottom] = net[right][bottom] +( fT*fL );
                net[left][top]     = net[left][top]     +( fB*fR );
                net[right][top]    = net[right][top]    +( fB*fL );
        }








/*
       int max = 1024;
        //int min = 0;

        float net[1024][1024];
        int i;
        int j;
        int size = 10000;

        double p[size][2]; //rows is number of particles. (first coloumn , second coloumn) is (x,y)
        float totalTime;

        //      cudaEvent_t start1, stop1;
        //        cudaEventCreate(&start1);
        //        cudaEventCreate(&stop1);
        //        float time;

        for(j=0; j < size;j++)
        {

                cudaEvent_t start1, stop1;
                cudaEventCreate(&start1);
                cudaEventCreate(&stop1);
                float time;




                //intialising particles.
                for( i = 0; i < size; i++)
                {
                   p[i][0]= ((float)rand()/(float)(RAND_MAX) * (float)max);
                   p[i][1]= ((float)rand()/(float)(RAND_MAX) * (float)max);
                }

                printf("\nvalue of P are %f, %f \n", p[size-1][0], p[size-1][1] );

                cudaEventRecord (start1,0);

                for ( i = 0; i < size; ++i)
                {
                        float x = p[i][0];
                        float y = p[i][1];

                        int left = (int)floor(x);
                        int right = left + 1;

                        int bottom = (int)floor(y);
                        int top = bottom +1;

                        float fL = x - left;
                        float fR = 1 - fL;

                        float fB = y - bottom;
                        float fT = 1 - fB;

                        net[left][bottom]  = net[left][bottom]  +( fT*fR );
                        net[right][bottom] = net[right][bottom] +( fT*fL );
                        net[left][top]     = net[left][top]     +( fB*fR );
                        net[right][top]    = net[right][top]    +( fB*fL );
                }
                cudaEventRecord (stop1,0);
                cudaEventSynchronize ( stop1 );
                cudaEventElapsedTime ( &time, start1, stop1 );
                printf("\n seconds: %f \n", time);

                cudaEventDestroy(start1);
                cudaEventDestroy(stop1);


                totalTime = totalTime+time;
        }

        printf("\nSeconds: %f \n", totalTime);
*/

        FILE *f = fopen("file.txt", "w");
        if (f == NULL)
        {
        printf("Error opening file!\n");
        exit(1);
        }

                for ( i = 0; i < 1023; ++i)
                {
                        for ( j = 0; j < 1023; j++)
                        {

                                fprintf (f,"%f ,",net[i][j] );
                        }

                        fprintf (f,"\n" );
                }

        fclose(f);


        return 0;
}



