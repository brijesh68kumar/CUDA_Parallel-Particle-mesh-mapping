/* Algo to so the weight distribution of 5000 particle on a
   grid of 64x64 */


//#include<conio.h>

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


#define funcCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            printf( "Failed to run stmt %d ", __LINE__);                       \
            printf( "Got CUDA error ...  %s \n", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)


//__device__ float floorf       (float  x);


__global__ void parMap(float *p, float *net, int grid)
{
        int rID= blockDim.y*blockIdx.y + threadIdx.y;
        int x,y, left, right, top, bottom;
        float fL,fR,fB,fT;

                x = p[rID*2];

                y = p[rID*2+1];

                left = (int)floorf(x);
                right = left + 1;

                bottom = (int)floorf(y);
                top = bottom +1;

                if (left>= grid||right>= grid||top>= grid||bottom>= grid)
                {
                        left=0;
                        right=1;
                        top=1;
                        bottom = 0;
                }

                fL = x - left;
                fR = 1 - fL;

                fB = y - bottom;
                fT = 1 - fB;

                net[grid*left + bottom]  = net[grid*left + bottom] +(fT*fR);
                net[grid*right + bottom] = net[grid*right + bottom]+(fT*fL);
                net[grid*left+ top]      = net[grid*left + top]    +(fB*fR);
                net[grid*right+ top]     = net[grid*right + top]   +(fB*fL);

}



// main function
int main(int argc, char *argv[])
{
        int grid = 1024, i, max = grid, par=1024, sizeGrid= grid*grid, sizePar=par*2;

        float netH[sizeGrid], pH[sizePar], *netD,  *pD;

        //netH =        (float )malloc(sizeof(float)*sizeGrid);
        //pH   =        (float )malloc(sizeof(float)*par*2);
        //intialising particles.

        for( i = 0; i < sizePar; i++)
                pH[i]= ((float)rand()/(float)(RAND_MAX) * (float)max);


        for(i=0;i<sizeGrid;i++)
                        netH[i]=0;

        // Allocating GPU memory
        funcCheck(hipMalloc((void **)&netD, sizeof(float)*sizeGrid));
        funcCheck(hipMalloc((void **)&pD, sizeof(float)*sizePar));

//    funcCheck(cudaMemcpy(netD, netH, grid*grid*sizeof(float), cudaMemcpyHostToDevice));
    funcCheck(hipMemcpy(pD,   pH,   sizePar*(sizeof(float)),     hipMemcpyHostToDevice));
    funcCheck(hipMemcpy(netD, netH, sizeGrid*(sizeof(float)),  hipMemcpyHostToDevice));


    // Initialize the grid and block dimensions
    dim3 dimBlock(32, 1);
    dim3 dimGrid((par/32) + 1, 1);


    //@@ Launch the GPU Kernel here
    parMap<<<dimGrid, dimBlock>>>(netD, pD, grid);

    hipError_t err1 = hipPeekAtLastError();

    // Copy the results in GPU memory back to the CPU
    funcCheck(hipMemcpy(netH, netD, sizeof(float)*sizeGrid, hipMemcpyDeviceToHost));


//!! if(x<0) stop print i
//!! denominator -- nan


        FILE *f = fopen("file.txt", "w");
        if (f == NULL)
        {
            printf("Error opening file!\n");
            exit(1);
        }

        float temp1=par/(grid*grid);

        for ( i = 0; i < grid*grid; ++i)
        {
            fprintf (f,"%f ",((netH[i])/temp1));
            if (i%grid==0)
                fprintf (f," \n" );
        }

        fclose(f);

        // Free the GPU memory
        funcCheck(hipFree(netD));
        funcCheck(hipFree(pD));

        free(netH);
        free(pH);

        return 0;
}

