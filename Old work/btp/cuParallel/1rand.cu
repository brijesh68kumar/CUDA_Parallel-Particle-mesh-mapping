// compile with: nvcc -arch=sm_20 -lcurand -o t89 t89.cu

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define SCALE 49000
#define DSIZE 5000
#define nTPB 256

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

__device__ float getnextrand(hiprandState *state){

  return (float)(hiprand_uniform(state));
}

__device__ int getnextrandscaled(hiprandState *state, int scale){

  return (int) scale * getnextrand(state);
}


__global__ void initCurand(hiprandState *state, unsigned long seed){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, 0, 0, &state[idx]);
}

__global__ void testrand(hiprandState *state, int *a1, int *a2){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

//    a1[idx] = getnextrandscaled(&state[idx], SCALE);
//    a2[idx] = getnextrandscaled(&state[idx], SCALE);
    a1[idx] = getnextrand(&state[idx]);
    a2[idx] = getnextrand(&state[idx]);

}

int main() {

    int *h_a1, *h_a2, *d_a1, *d_a2;
    hiprandState *devState;

    h_a1 = (int *)malloc(DSIZE*sizeof(int));
    if (h_a1 == 0) {printf("malloc fail\n"); return 1;}
    h_a2 = (int *)malloc(DSIZE*sizeof(int));
    if (h_a2 == 0) {printf("malloc fail\n"); return 1;}
    hipMalloc((void**)&d_a1, DSIZE * sizeof(int));
    hipMalloc((void**)&d_a2, DSIZE * sizeof(int));
    hipMalloc((void**)&devState, DSIZE * sizeof(hiprandState));
    cudaCheckErrors("cudamalloc");



     initCurand<<<(DSIZE+nTPB-1)/nTPB,nTPB>>>(devState, 1);
     hipDeviceSynchronize();
     cudaCheckErrors("kernels1");
     testrand<<<(DSIZE+nTPB-1)/nTPB,nTPB>>>(devState, d_a1, d_a2);
     hipDeviceSynchronize();
     cudaCheckErrors("kernels2");
     hipMemcpy(h_a1, d_a1, DSIZE*sizeof(int), hipMemcpyDeviceToHost);
     hipMemcpy(h_a2, d_a2, DSIZE*sizeof(int), hipMemcpyDeviceToHost);
     cudaCheckErrors("cudamemcpy");
     printf("1st returned random value is %d\n", h_a1[0]);
     printf("2nd returned random value is %d\n", h_a2[0]);

     for (int i=1; i< DSIZE; i++){
       if (h_a1[i] != h_a1[0]) {
         printf("mismatch on 1st value at %d, val = %d\n", i, h_a1[i]);
         return 1;
         }
       if (h_a2[i] != h_a2[0]) {
         printf("mismatch on 2nd value at %d, val = %d\n", i, h_a2[i]);
         return 1;
         }
       }
     printf("thread values match!\n");

}
