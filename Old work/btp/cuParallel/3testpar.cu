/* Algo to so the weight distribution of 5000 particle on a
   grid of 64x64 */


//#include<conio.h>

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>

using namespace std;

//__device__ float floorf       (float  x);


__global__ void parMap(float *p, float *net, int grid)
{
        int rID= blockDim.y*blockIdx.y + threadIdx.y;
        int x,y, left, right, top, bottom;
        float fL,fR,fB,fT;

                x = p[rID*2];

                y = p[rID*2+1];

//				printf("%d %f %f       ",rID,x,y);
//		cout<<x<<"  ";
                left = (int)floorf(x);
                right = left + 1;

                bottom = (int)floorf(y);
                top = bottom +1;

                if (left>= grid||right>= grid||top>= grid||bottom>= grid)
                {
                        left=0;
                        right=1;
                        top=1;
                        bottom = 0;
                }

                fL = x - left;
                fR = 1 - fL;

                fB = y - bottom;
                fT = 1 - fB;

                net[grid*left + bottom]  = net[grid*left + bottom] +(fT*fR);
                net[grid*right + bottom] = net[grid*right + bottom]+(fT*fL);
                net[grid*left+ top]      = net[grid*left + top]    +(fB*fR);
                net[grid*right+ top]     = net[grid*right + top]   +(fB*fL);

}



// main function
int main(int argc, char *argv[])
{
        int grid = 1024, i, max = grid, par=145623, sizeGrid= grid*grid, sizePar=par*2;

        	float* netH;
		float* pH;
		float* netD;
		float*  pD;

        netH =        (float*)malloc(sizeof(float)*sizeGrid);
        pH   =        (float*)malloc(sizeof(float)*par*2);
        //intialising particles.

      for( i = 0; i < sizePar; i++)
               pH[i]= ((float)rand()/(float)(RAND_MAX) * (float)max);

	printf("particle initialised \n "); 

	
        for(i=0;i<sizeGrid;i++)
                        netH[i]=0;
        printf("Grid initialised \n ");

        for(i=0;i<10;i++)
               //printf("%f, %f   ", netH[i], pH);
	cout<<netH[i]<<" "<<pH[i]<<" , ";


        // Allocating GPU memory
        hipMalloc((void **)&netD, sizeof(float)*sizeGrid);
        hipMalloc((void **)&pD, sizeof(float)*sizePar);

        printf("Cuda memory allocated \n ");

//    funcCheck(cudaMemcpy(netD, netH, grid*grid*sizeof(float), cudaMemcpyHostToDevice));
    hipMemcpy(pD,   pH,   sizePar*(sizeof(float)),     hipMemcpyHostToDevice);
    hipMemcpy(netD, netH, sizeGrid*(sizeof(float)),  hipMemcpyHostToDevice);

        printf("Data cpy to gpu \n \n ");

    // Initialize the grid and block dimensions
    dim3 dimBlock(32, 1);
    dim3 dimGrid(par/32,1);


    //@@ Launch the GPU Kernel here
    parMap<<<dimGrid, dimBlock>>>(netD, pD, grid);

    hipError_t err1 = hipPeekAtLastError();

        printf("Data back to CPU \n \n ");



    // Copy the results in GPU memory back to the CPU
    hipMemcpy(netH, netD, sizeof(float)*sizeGrid, hipMemcpyDeviceToHost);


//!! if(x<0) stop print i
//!! denominator -- nan

		/*
        FILE *f = fopen("file.txt", "w");
        if (f == NULL)
        {
            printf("Error opening file!\n");
            exit(1);
        }*/

        //  float temp1=par/(sizeGrid);

        for ( i = 0; i < 100; ++i)
        {
		
		//	printf("%f ",netH[i]);
		cout<< netH[i]<< "  ";
            //fprintf (f,"%f ",((netH[i])/temp1));
            if (i%grid==0)
		{
			printf("\n");
                	//fprintf (f," \n" );
		}
        }

        //fclose(f);

        // Free the GPU memory
        hipFree(netD);
        hipFree(pD);
		
//        free(netH);
//        free(pH);
		
        return 0;
}


