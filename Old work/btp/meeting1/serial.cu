/* Algo to so the weight distribution of 5000 particle on a 
   grid of 64x64 */


//#include<conio.h>

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>


//function to calculate the contribution on each coordinate.


// main function
int main(int argc, char *argv[])
{
	hipEvent_t start_i, stop_i;
	float time_i;
	
	
	hipEventCreate(&start_i);
	hipEventCreate(&stop_i);
	

	int max = 64;
//	int min = 0;

	float net[64][64];
	int i;
	int j;

	float p[20500][2]; //rows is number of particles. (first coloumn , second coloumn) is (x,y) 

        printf("Till initialisation\n");



	//intialising particles.
	for( i = 0; i < 20500; i++)
	{
		p[i][0]= ((float)rand()/(float)(RAND_MAX) * (float)max);
		p[i][1]= ((float)rand()/(float)(RAND_MAX) * (float)max);
	}

        printf("elements are decleared\n");

	hipEventRecord(start_i,0);
	
	for ( i = 0; i < 20450; ++i)
	{
		float x = p[i][0];
		float y = p[i][1];

		int left = (int)floor(x);
		int right = left + 1;

		int bottom = (int)floor(y);
		int top = bottom +1;

		float fL = x - left;
		float fR = 1 - fL;

		float fB = y - bottom;
		float fT = 1 - fB;

		net[left][bottom] 	= 	net[left][bottom]	+( fT * fR ) ;
		net[right][bottom] 	= 	net[right][bottom]	+( fT * fL ) ;
		net[left][top]		=	net[left][top]		+( fB * fR ) ;
		net[right][top]		=	net[right][top]		+( fB * fL ) ;
	}
	hipEventRecord(stop_i,0);
	hipEventElapsedTime( &time_i, start_i,stop_i );
	printf("\n Total processing time: %f \n", time_i );


	FILE *f = fopen("file.txt", "w");
	if (f == NULL)
	{
    	printf("Error opening file!\n");
    	exit(1);
	}

		for ( i = 0; i < 64; ++i)
		{
			for ( j = 0; j < 64; j++)
			{

				fprintf (f,"%f ,",net[i][j] );
			}

			fprintf (f,"\n" );
		}



	fclose(f);
	hipEventDestroy(start_i);
	hipEventDestroy(stop_i);



	return 0;
}

