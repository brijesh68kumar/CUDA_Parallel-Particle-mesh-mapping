
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <sys/time.h>
#include "time.h"

using namespace std;

__global__ void parMap(float *pD, float *netD, int grid)
{
        unsigned int rID= blockDim.x*blockIdx.x + threadIdx.x;
        int left, right, top, bottom;
        float x,y, fL,fR,fB,fT;

        x = pD[rID*2];
        y = pD[rID*2+1];

        left = (int)floorf(x);
        right = left + 1;
        bottom = (int)floorf(y);
        top = bottom +1;

        if (left>= grid||right>= grid||top>= grid||bottom>= grid){
            left=0;
            right=1;
            top=1;
            bottom = 0;
            x=0.500000;
            y=0.500000;
        }

        fL = x - left;
        fR = 1 - fL;
        fB = y - bottom;
        fT = 1 - fB;

        netD[grid*left + bottom]  = netD[grid*left + bottom] +(fT*fR);
        netD[grid*right + bottom] = netD[grid*right + bottom]+(fT*fL);
        netD[grid*left+ top]      = netD[grid*left + top]    +(fB*fR);
        netD[grid*right+ top]     = netD[grid*right + top]   +(fB*fL);
}

int main(int argc, char *argv[])
{
	//--------------------Declaring Variables-------------------------
        int grid = 1024, i, j, lp=1,max = grid, sizeGrid= grid*grid;
        unsigned int par = 160000, loop=2000, sizePar = 2*par;
        float t_i=0.0, t_mc_h2d=0.0, t_mc_d2h=0.0, t_pl=0.0, ti=0.0, tmc_h2d=0.0, tpl=0.0;
        hipEvent_t s_i, e_i, s_mc_h2d, e_mc_h2d, s_mc_d2h, e_mc_d2h, s_pl, e_pl;
        float *netH, *pH, *netD,  *pD;
		//___Time flags___	
        hipEventCreate(&s_i);
        hipEventCreate(&e_i);
		hipEventCreate(&s_mc_h2d);
		hipEventCreate(&e_mc_h2d);
        hipEventCreate(&s_mc_d2h);
        hipEventCreate(&e_mc_d2h);
		hipEventCreate(&s_pl);
		hipEventCreate(&e_pl);
		//________________
	//-----------------------------------------------------------------

	//--------------------Initializing data--------------------------
		//___start clock___.
        hipEventRecord(s_i,0);

			//___CPU Memory allocation___
			netH =        (float*)malloc(sizeof(float)*sizeGrid);
			pH   =        (float*)malloc(sizeof(float)*sizePar);
			//___________________________
			//___initializing grid___
			for(i=0;i< grid;i++)
                for(j=0;j< grid;j++)
                        netH[grid*i+j]=0.0;
			//___________________________
			//___Random particle position___				
			for( i = 0; i < sizePar; i++)
                pH[i]= ((float)rand()/(float)(RAND_MAX) * (float)(max-1));
			//___________________________

        hipEventRecord( e_i,0 );
        hipEventSynchronize( e_i );
        hipEventElapsedTime( &ti, s_i, e_i);
		//___________________________
	//-----------------------------------------------------------------
	//--------------------GPU memory allocation for grid--------------------------
		//___start clock___.
		hipEventRecord(s_mc_h2d,0);		
			
			//___GPU memory allocation___
			hipMalloc( (void **)&netD, sizeof(float)*sizeGrid);
			//___________________________
			//___Data Transfer___
			hipMemcpy(netD, netH, sizeGrid*(sizeof(float)),  hipMemcpyHostToDevice);
			//___________________

		hipEventRecord( e_mc_h2d,0 );
        hipEventSynchronize( e_mc_h2d );
        hipEventElapsedTime( &tmc_h2d, s_mc_h2d, e_mc_h2d);
		t_mc_h2d+=tmc_h2d; //calculating time
		//___________________________
	//----------------------------------------------------------------------------

	//--------------------Parallel implementation --------------------------
	for(lp=1;lp<loop;lp++){   
	//___perticle data transfer___
		hipEventRecord(s_mc_h2d,0);
			//___Allocating GPU memory___
			hipMalloc( (void **)&pD, sizeof(float)*sizePar);
			//___Memory transfer from CPU to GPU___
			hipMemcpy( pD,   pH,   sizePar*(sizeof(float)), hipMemcpyHostToDevice);
		hipEventRecord( e_mc_h2d,0 );
		hipEventSynchronize( e_mc_h2d );
		hipEventElapsedTime( &tmc_h2d, s_mc_h2d, e_mc_h2d);
	//___Launching threads___		
		hipEventRecord( s_pl,0 );
			//___thread dimentions___
			dim3 dimBlock(192);
			dim3 dimGrid((par/192));
			//___kernel Launch___
			parMap<<<dimGrid, dimBlock>>>(pD, netD, grid);			
		hipEventRecord( e_pl,0 );
		hipEventSynchronize( e_pl );
		hipEventElapsedTime( &tpl, s_pl, e_pl);
	//___Time keeing___
		t_i+=ti;
		t_mc_h2d+=tmc_h2d;
		t_pl+=tpl;	
	}

	//___copy results from GPU to CPU___
        hipEventRecord( s_mc_d2h,0 );
			hipMemcpy(netH, netD, sizeof(float)*sizeGrid, hipMemcpyDeviceToHost);
        hipEventRecord( e_mc_d2h,0 );
        hipEventSynchronize( e_mc_d2h );
        hipEventElapsedTime( &t_mc_d2h, s_mc_d2h, e_mc_d2h);
	//---------------------------------------------------------------------
	//--------------------Saving result in file --------------------------
		//___Opening file___
		FILE *f = fopen("file.txt", "w");
		par*=loop;
        if (f == NULL){
            printf("Error opening file!\n");
            exit(1);
        }

        float avg= par/(max*max);

        for ( i = 0; i < sizeGrid; ++i){	
			fprintf (f,"%f ",((netH[i])/avg))  ;
            if (i%grid==(grid-1))
                fprintf (f," \n" );
        }

        fclose(f);
	//----------------------------------------------------------------------------
		
        printf("\n\nGrid size: \t\t%d \n particle:\t %d\n", grid,par);
        printf("\nInitialisation time:\t%f \n", t_i);
        printf("\nMemory Copy H 2 D:\t%f \n", t_mc_h2d);
        printf("\nMemory Copy D 2 H:\t%f \n", t_mc_d2h);
        printf("\nProcessing time:\t%f \n\n", t_pl);

        //___destroy events___
        hipEventDestroy(s_i);
        hipEventDestroy(e_i);    
		hipEventDestroy(s_mc_h2d);
		hipEventDestroy(e_mc_h2d);
        hipEventDestroy(s_pl);
        hipEventDestroy(e_pl);    
		hipEventDestroy(s_mc_d2h);
		hipEventDestroy(e_mc_d2h);

        // Free memory
        hipFree(netD);
        hipFree(pD);
        free(netH);
        free(pH);

        return 0;
}

